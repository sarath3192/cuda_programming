#include <iostream>
#include <hip/hip_runtime.h>

int main() {
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0) {
        std::cout << "No CUDA-compatible GPU detected.\n";
        return 0;
    }

    std::cout << "CUDA Devices:\n";
    for (int i = 0; i < deviceCount; ++i) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);

        std::cout << "Device " << i << ": " << prop.name << "\n";
        std::cout << "  Compute Capability: " << prop.major << "." << prop.minor << "\n";
        std::cout << "  Total Global Memory: " << (prop.totalGlobalMem >> 20) << " MB\n";
        std::cout << "  Multiprocessors: " << prop.multiProcessorCount << "\n";
        std::cout << "  Clock Rate: " << prop.clockRate / 1000 << " MHz\n\n";
    }

    return 0;
}
