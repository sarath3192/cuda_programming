#include <iostream>
#include <hip/hip_runtime.h>

void checkCudaDevice() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0) {
        std::cout << "CUDA device not found!" << std::endl;
        return;
    }

    int deviceId;
    hipGetDevice(&deviceId);

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, deviceId);

    std::cout << "Using CUDA device: " << deviceProp.name << std::endl;
    std::cout << "Total Memory: " << deviceProp.totalGlobalMem / (1024 * 1024) << " MB" << std::endl;
}

int main() {
    checkCudaDevice();
    return 0;
}
